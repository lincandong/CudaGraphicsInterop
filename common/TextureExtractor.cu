#include "hip/hip_runtime.h"
#include "TextureExtractor.h"
#include <glm/glm.hpp>
#include <vector>
#include <string>

hipChannelFormatDesc GetChannelDesc(TextureFormat format) {
    switch (format)
    {
        case TextureFormat::RGBA8:
            return hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
        case TextureFormat::RGB10A2:
            return hipCreateChannelDesc(10, 10, 10, 2, cudaChannelFormatKindUnsignedNormalized1010102);
        case TextureFormat::RGBA32:
            // return hipCreateChannelDesc(10, 10, 10, 2, cudaChannelFormatKindUnsignedNormalized1010102);
            return hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
        default:
            throw std::runtime_error("Invalid channel format");
    }
}

size_t GetChannelSize(TextureFormat format) {
    switch (format)
    {
        case TextureFormat::RGBA8:
            return 4;
        case TextureFormat::RGB10A2:
            return 4;
        case TextureFormat::RGBA32:
            return 4 * 4;
        default:
            throw std::runtime_error("Invalid channel format");
    }
}

TextureExtractor::TextureExtractor(std::string& resName, TextureFormat textureFormat, int textureWidth, int textureHeight, int textureBytes)
{
    resourceName = resName;
    width = textureWidth;
    height = textureHeight;
    format = textureFormat;
    totalBytes = textureBytes;
}

std::vector<glm::vec3> TextureExtractor::extractTextureData() {
    if (!initialized) {
        return {};
    }

    std::vector<glm::vec3> result(width * height);

    // Allocate device memory
    glm::vec3* d_rgb = nullptr;
    CUDA_CHECK(hipMalloc(&d_rgb, width * height * sizeof(glm::vec3)));

    hipArray_t cuArray;
    hipChannelFormatDesc channelDesc = GetChannelDesc(format);
    size_t channelSize = GetChannelSize(format);
    hipMallocArray(&cuArray, &channelDesc, width, height);
    CUDA_CHECK(hipMemcpyToArray(cuArray, 0, 0, devicePtr, width * height * channelSize, hipMemcpyDeviceToDevice));

    hipResourceDesc resDesc = {};
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // hipArray_t baseArray;
    // CUDA_CHECK(hipGetMipmappedArrayLevel(&baseArray, mipArray, 0));
    // resDesc.resType = hipResourceTypeArray;
    // resDesc.res.array.array = baseArray;

    // resDesc.resType = hipResourceTypeMipmappedArray;
    // resDesc.res.mipmap.mipmap = mipArray;

    // resDesc.resType = hipResourceTypePitch2D;
    // resDesc.res.pitch2D.desc = hipCreateChannelDesc(10, 10, 10, 2, cudaChannelFormatKindUnsignedNormalized1010102);
    // resDesc.res.pitch2D.devPtr = devicePtr;
    // resDesc.res.pitch2D.width = width;
    // resDesc.res.pitch2D.height = height;
    // // resDesc.res.pitch2D.pitchInBytes = rowPitch;
    // // resDesc.res.pitch2D.pitchInBytes = (width+2) * sizeof(uint32_t);
    // resDesc.res.pitch2D.pitchInBytes = (width) * sizeof(uint32_t);
    // // resDesc.res.pitch2D.pitchInBytes = (width+2);

    // resDesc.resType = hipResourceTypeLinear;
    // resDesc.res.linear.desc = hipCreateChannelDesc(10, 10, 10, 2, cudaChannelFormatKindUnsignedNormalized1010102);
    // resDesc.res.linear.devPtr = devicePtr;
    // // resDesc.res.linear.sizeInBytes = totalBytes;
    // resDesc.res.linear.sizeInBytes = width * height * sizeof(uint32_t);

    hipTextureDesc texDesc = {};
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;     // No filtering
    texDesc.readMode = hipReadModeElementType;   // Read raw values
    texDesc.normalizedCoords = 0;                 // Use integer coordinates for fetch

    hipTextureObject_t texObj = 0;
    CUDA_CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr));

    // Launch conversion kernel
    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
    unpackRGB32Kernel<<<blocks, threads>>>(texObj, d_rgb, width, height);

    // if (packed)
    // {
    //     unpackRGB10A2PackedKernel<<<blocks, threads>>>((uint32_t*)devicePtr, d_rgb, width, height);
    // }
    // else
    // {
    //     unpackRGB32Kernel<<<blocks, threads>>>(texObj, d_rgb, width, height);
    // }
    
    // Copy result back to host
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(result.data(), d_rgb, width * height * sizeof(glm::vec3), hipMemcpyDeviceToHost));
    
    // Clean up device memory
    hipFree(d_rgb);
    
    return result;
}

__global__ void unpackRGB32Kernel(hipTextureObject_t texObj, glm::vec3* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = y * width + x;
        float4 pixel = tex2D<float4>(texObj, x, y);
        output[idx] = glm::vec3(pixel.x, pixel.y, pixel.z);
    }
}

__global__ void unpackRGB10A2PackedKernel(uint32_t* data, glm::vec3* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int inIndex = y * width + x;
        int outIndex = y * width + x;
        uint32_t pixel = data[inIndex];
        output[outIndex] = glm::vec3(
            (pixel & 0x3FF) / 1023.0f,
            ((pixel >> 10) & 0x3FF) / 1023.0f,
            ((pixel >> 20) & 0x3FF) / 1023.0f
        );
    }
}
